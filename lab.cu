/*
 ============================================================================
 Name        : lab.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <fstream>

//#include <stdio.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

struct Params
{
	int length;
	int time;
	float dx;
	float dt;
	float startT;
	float endT;
};

void initialize(float **data, Params inParam)
{
	for (unsigned i = 0; i < inParam.time; ++i)
		data[i] = new float[inParam.length];
}

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *OldData,float *NewData, Params inParams) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
			if(idx == 0)
				NewData[idx] = inParams.startT;
			else if (idx == inParams.length - 1 )
				NewData[idx] = inParams.endT * inParams.dt + OldData[idx];
			else
			{
				NewData[idx] = ((OldData[idx+1] - 2 * OldData[idx] + OldData[idx-1])* inParams.dt)/inParams.dx * inParams.dx + OldData[idx];
			}

}

/**
 * Host function that copies the data and launches the work on GPU
 */
void gpuReciprocal(float *data, Params inParams)
{
	float *rc = new float[inParams.length];
	float *gpuOldData;
	float *gpuNewData;

	hipEvent_t GPUstart, GPUstop;
		float GPUtime = 0.0f;


		hipEventCreate(&GPUstart);
		hipEventCreate(&GPUstop);

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuOldData, sizeof(float)*inParams.length));
	CUDA_CHECK_RETURN(hipMemcpy(gpuOldData, rc, sizeof(float)*inParams.length, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuNewData, sizeof(float)*inParams.length));
	CUDA_CHECK_RETURN(hipMemcpy(gpuNewData, data, sizeof(float)*inParams.length, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 26;
	const int blockCount = 1;
	for(int i = 0; i < inParams.time; ++i)
	{
		hipEventRecord(GPUstart, 0);
		if (i % 2 == 0)
		{
			reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuOldData, gpuNewData, inParams);
			hipEventRecord(GPUstop, 0);
			CUDA_CHECK_RETURN(hipMemcpy(rc, gpuNewData, sizeof(float)*inParams.length, hipMemcpyDeviceToHost));

		}
			else
			{

			reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuNewData, gpuOldData, inParams);
			hipEventRecord(GPUstop, 0);
		    CUDA_CHECK_RETURN(hipMemcpy(rc, gpuOldData, sizeof(float)*inParams.length, hipMemcpyDeviceToHost));
			}

		hipEventSynchronize(GPUstop);

		std::cout << i <<": ";
		for(int i =0; i < inParams.length; ++i)
						std::cout << rc[i] << " ";
		std::cout << std::endl;
		float temp;
		hipEventElapsedTime(&temp,GPUstart, GPUstop);
		GPUtime += temp;

	}
	printf("GPU time = %.3f ms\n",GPUtime);
}

void initializeData(float* data, unsigned size,  float defaultValue)
{
	for(int i = 0; i < size; ++i)
		data[i] = defaultValue;
}



void print(float **a, unsigned h, unsigned w)
{
	for(int i = 0; i < h; ++i)
	{
		std::cout << i << ": ";
		for(int j = 0; j < w; ++j)
			std::cout << a[i][j] << " ";
		std::cout << std::endl;
	}

}


float **cpuReciprocal(float **data,Params inParams)
{
	float CPUstart, CPUstop;
	float CPUtime = 0.0f;

	float **cpuResult = new float*[inParams.time];
	initialize(cpuResult, inParams);

	float * currentData = cpuResult[0];
    initializeData(currentData, inParams.length, 0);


     currentData[0] = inParams.startT;

	for(int i =1; i < inParams.time; ++i)
	{
		cpuResult[i][0] = inParams.startT;
		for(int j = 1; j < inParams.length - 1; ++j)
		{
			cpuResult[i][j] = ((cpuResult[i-1][j+1] - 2 * cpuResult[i-1][j] + cpuResult[i-1][j-1])* inParams.dt)/inParams.dx * inParams.dx + cpuResult[i-1][j];
		}
		cpuResult[i][inParams.length-1] = inParams.endT * inParams.dt + cpuResult[i-1][inParams.length-1];
	}

	return cpuResult;
}






int main(void)
{
	Params mainParams;
	mainParams.length = 13;
	mainParams.time = 20;
	mainParams.dx = 0.5;
	mainParams.dt = 0.1;
	mainParams.startT = 0.0;
	mainParams.endT = 5.0;

	float **data;
	data = new float*[mainParams.time];
	initialize(data, mainParams);

	std::cout << "CPU" << std::endl;
	float **recCpu = cpuReciprocal(data, mainParams);
	print(recCpu, mainParams.time, mainParams.length);

	std::cout << "GPU" << std::endl;
	float *dataTwo;
	dataTwo = new float[mainParams.length];
	gpuReciprocal(dataTwo, mainParams);
	//print(recGpu, mainParams.time, mainParams.length);
//	float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
//	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);

	/* Verify the results */
	//std::cout<<"gpuSum = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;

	/* Free memory */
	//delete[] data;
	//delete[] recCpu;
	//delete[] recGpu;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

